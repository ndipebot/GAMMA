#include "hip/hip_runtime.h"

/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <deviceHeader.h>
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <algorithm>

__constant__ double parCoords[24];
__constant__ double parCoordsSurf[72];
__constant__ double parCoords2D[8];
__constant__ int mapIndex[64];

__global__ void updateMass(const double* __restrict__ thetaN, const double* __restrict__ eleMat, const double* __restrict__ eleNodeCoords,
		double* globMass, const int* __restrict__ eleNodes, const int* __restrict__ nUniId, const int numEl, const int nn, const int numElAct);

__global__ void initializeStiffness(const double* __restrict__ eleMat, const double* __restrict__ eleNodeCoords, double* eleStiffness,
		const int* __restrict__ eleNodes, const int numEl, const int nn);

__global__ void getInternalForce(double* globRHS, double* __restrict__ eleStiffness, double* __restrict__ thetaN, 
	const int* __restrict__ eleNodes, const int* __restrict__ nUniId, const int numEl, const int nn, const int numElAct);

__global__ void massReduce(double* __restrict__ globMass, const int nn, const int rhsCount);

__global__ void interForceReduce(double* __restrict__ globRHS, const int nn);

__global__ void applyFlux(const double* __restrict__ surfFlux, const int* __restrict__ surfPlane, const double* __restrict__ thetaN, const int* __restrict__ surfIndx,
	const double* __restrict__ surfNodeCoords, const double* __restrict__ eleNodeCoords, const int* __restrict__ surfNodes, const int* __restrict__ surfElemBirth,
	double* rhs, int numSurf, double ambient, double abszero, double tool0, double tool1, double tool2, int laserState, int numSurfAct, double sigma, int numEl, int nn);

__global__ void massFlux(double* rhs, const int nn, int rhsCount, double* globalRhs);

__global__ void advanceTime(double* thetaN, const double* __restrict__ globalMass, const double* __restrict__ rhs,const int* __restrict__ birthNodes, double dt, int numActNodes);

__global__ void prescribeDirichlet(double* thetaN, const int* __restrict__ fixedNodes, const double* __restrict__ fixedNodeVals, int numFixed);

void compareMass(elementData& elemData, vector<double>& Mvec) {
	int nn = Mvec.size();
	int base = 0;
	for (int i = 0; i < nn; i++) {
		if (abs(elemData.globMass[base + i] - Mvec[base + i]) > 0.000001) {
			std::cout << "Mismatch found on node: "<<i<<", GPU: "<< elemData.globMass[base + i] << ", CPU: "<< Mvec[base + i]<<std::endl;
		}
	}
	std::cout << "check passed!" << std::endl;
}

void compareStiff(elementData& elemData, vector<Element*>& elementList) {
	int numEl = elementList.size();
	for (const auto & elem : elementList) {
		int eID = &elem - &elementList[0];
		for (int i = 0; i < 36; ++i) {
			if (abs(elemData.eleStiffness[eID + i*numEl] - elem->stiffMatrix_[i]) > 0.0001) {
				std::cout << "Mismatch found on element: " << eID << ", GPU: " << elemData.eleStiffness[eID + i*numEl] << ", CPU: " << elem->stiffMatrix_[i] << std::endl;
			}
		}
	}
}

void compareIntForce(elementData& elemData, vector<double>& rhs) {
	int nn = rhs.size();
	int base = 0;
	for (int i = 0; i < nn; i++) {
		std::cout << ", GPU: " << elemData.globRHS[base + i]<< ", CPU: " << rhs[base + i] << std::endl;
		if (abs(elemData.globRHS[base + i] - rhs[base + i]) > 0.000001) {
			std::cout << "Mismatch found on node: " << i << ", GPU: " << elemData.globRHS[base + i] << ", CPU: " << rhs[base + i] << std::endl;
		}
	}
	std::cout << "check passed!" << std::endl;
}

void compareFlux(elementData& elemData, vector<double>& rhs) {
	int nn = rhs.size();
	int base = 0;
	for (int i = 0; i < nn; i++) {
		if (abs(elemData.globRHS_Surf[base + i] - rhs[base + i]) > 0.000001) {
			std::cout << "Mismatch found on node: " << i << ", GPU: " << elemData.globRHS_Surf[base + i] << ", CPU: " << rhs[base + i] << std::endl;
		}
	}
	std::cout << "check passed!" << std::endl;
}

void compareTemp(elementData& elemData, vector<double>& thetaN ) {
	int base = 0;
	for (int i = 0; i < elemData.nn; i++) {
		if (abs(elemData.thetaN[base + i] - thetaN[base + i]) > 0.000001) {
			std::cout << "Mismatch found on node: " << i << ", GPU: " << elemData.thetaN[base + i] << ", CPU: " << thetaN[base + i] << std::endl;
		}
	}
	std::cout << "check passed!" << std::endl;
}

void createDataOnDevice(DomainManager*& domainMgr, elementData& elemData, HeatSolverManager*& heatMgr) {
    //Start recording time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Start record
    hipEventRecord(start, 0);

	/*
	* ELEMENTS
	*/

	int numEl = domainMgr->nel_;
	int nn = domainMgr->nn_;


	elemData.initTemp = heatMgr->initTheta_;
	elemData.nn = nn;
	elemData.numEl = numEl;
	elemData.dt = heatMgr->dt_;

	//Element data
	elemData.eleNodes.resize(8*numEl);
	elemData.eleNodeCoords.resize(24*numEl);
	elemData.eleMat.resize(6*numEl);
	elemData.eleBirthTimes.resize(numEl);
	elemData.thetaN.resize(nn, elemData.initTemp);

	elemData.eleStiffness.resize(numEl*36);


	elemData.nUniId.resize(numEl * 8);
	vector<int> nodeInd;
	nodeInd.resize(nn,0);
	int eleRowCnt = 0;

	// set element values
	for(const auto & elem : domainMgr->elementList_) {
		 int * nodes = elem->nID_;
		 int eID = &elem - &domainMgr->elementList_[0]; // grab current element id


		 for(int i = 0; i < 8; ++i) {
			 //assign nodes to device vector in coalesced manner
			 elemData.eleNodes[numEl*i + eID] = nodes[i];
			 elemData.nUniId[numEl*i + eID] = nodeInd[nodes[i]]++;
			 eleRowCnt = std::max(eleRowCnt, nodeInd[nodes[i]]);

			 //grab coalesced nodal coordinates while you're at it
			 for(int j = 0; j < 3; ++j)
				 elemData.eleNodeCoords[3*numEl*i + j*numEl + eID] = domainMgr->coordList_[nodes[i]*3+j];
		 }
		//material properties
		elemData.eleMat[eID] = elem->rho_;
		elemData.eleMat[numEl + eID] = elem->solidus_;
		elemData.eleMat[2*numEl + eID] = elem->liquidus_;
		elemData.eleMat[3*numEl + eID] = elem->latent_;
		elemData.eleMat[4*numEl + eID] = elem->cp_;
		elemData.eleMat[5*numEl + eID] = elem->cond_;

	    //element birth times
		elemData.eleBirthTimes[eID] = elem->birthTime_;
	}
	elemData.rhsCountEle = eleRowCnt;
	//-----------------------  END OF ELEMENT -----------------------------------------------///

	/*
	*  SURFACES
	*/
	// Surface ninjutsu
	vector<double> tempFlux;
	vector<int> tempNodes;
	vector<double> tempCoords;

	//grab static surface
	for (auto const surfID : heatMgr->heatBCManager_->activeSurfaces_) {
		Surface surf = heatMgr->heatBCManager_->staticSurfList_[surfID];
		elemData.boundSurfBirthTime.push_back(surf.birthTime_);
		elemData.boundSurfDeathTime.push_back(surf.deathTime_);
		elemData.surfPlane.push_back(surf.plane_);
		elemData.surfBirthElem.push_back(surf.birthElement_);

		for (int i = 0; i < 4; ++i)
			tempNodes.push_back(surf.surfaceNodes_[i]);

		for (int i = 0; i < 5; ++i)
			tempFlux.push_back(surf.flux[i]);

		for (int i = 0; i < 8; ++i)
			tempCoords.push_back(surf.mappedCoords[i]);
	}

	//grab birthSurfaces
	for (auto const surf : heatMgr->heatBCManager_->surfaceList_) {
		if (surf.isFlux_) {
			elemData.boundSurfBirthTime.push_back(surf.birthTime_);
			elemData.boundSurfDeathTime.push_back(surf.deathTime_);
			elemData.surfPlane.push_back(surf.plane_);
			elemData.surfBirthElem.push_back(surf.birthElement_);

			for (int i = 0; i < 4; ++i)
				tempNodes.push_back(surf.surfaceNodes_[i]);

			for (int i = 0; i < 5; ++i)
				tempFlux.push_back(surf.flux[i]);

			for (int i = 0; i < 8; ++i)
				tempCoords.push_back(surf.mappedCoords[i]);
		}
	}

	//get number of surfaces in total
	elemData.numSurf = elemData.boundSurfBirthTime.size();
	int numSurf = elemData.numSurf;
	elemData.surfNodes.resize(4 * numSurf, 0); // nodes
	elemData.surfIndx.resize(4*numSurf, 0);  // surface location
	elemData.surfNodeCoords.resize(8 * numSurf, 0);
	elemData.surfFlux.resize(5 * numSurf, 0); // flux contains [flux, flux variable], flux is 0 or 1

											  //count index nodes for rhs vector
	vector<int> nodeIndex(nn, 0);
	int maxCount = 0;

	//order surface nodes, nodal coordinates, and flux for coalesced accesses
	for (int i = 0; i < numSurf; ++i) {
		for (int j = 0; j < 4; ++j) {
			//nodes
			int nodeID = tempNodes[4 * i + j];
			elemData.surfNodes[j*numSurf + i] = nodeID;
			elemData.surfIndx[i*4 + j] = nodeIndex[nodeID]++;
			maxCount = std::max(maxCount, nodeIndex[nodeID]);
			//coordinates
			for (int k = 0; k < 2; ++k)
				elemData.surfNodeCoords[2 * numSurf*j + k*numSurf + i] = tempCoords[i * 8 + j*2 + k];
		}

		//flux
		for (int j = 0; j < 5; ++j)
			elemData.surfFlux[j*numSurf + i] = tempFlux[i * 5 + j];
	}

	elemData.rhsCount = maxCount;

	elemData.ambient = domainMgr->meshObj_->Rambient_;
	elemData.abszero = domainMgr->meshObj_->Rabszero_;
	elemData.sigma = domainMgr->meshObj_->Rboltz_;

	elemData.globMass.resize(nn * 12);
	elemData.globRHS.resize(nn * 12);
	elemData.globRHS_Surf.resize(nn * maxCount);


	//------------- Fixed Nodes --------------------------
	elemData.numFixed = heatMgr->heatBCManager_->fixedNodeIDs_.size();
	elemData.fixedNodes = heatMgr->heatBCManager_->fixedNodeIDs_.data();
	elemData.fixedValues = heatMgr->heatBCManager_->fixedNodeVals_.data();

	//------------------------birth Nodes
	for(int i = 0; i < domainMgr->activeNodes_.size(); ++i) {
		elemData.birthNodes.push_back(domainMgr->activeNodes_[i]);
		elemData.birthNodeTimes.push_back(0.0);
	}

	for(int i = 0; i < domainMgr->birthNodes_.size(); ++i) {
		elemData.birthNodes.push_back(domainMgr->birthNodes_[i]);
		elemData.birthNodeTimes.push_back(domainMgr->birthNodeTimes_[i]);
	}

	//----------------------------------------------------

	//Alocate and Copy
	AllocateDeviceData(elemData);
	CopyToDevice(elemData);

	//move parametric coordinates to local memory

	vector<double> coords(24);

	vector<double> coeff {-1.0, -1.0, -1.0,
							1.0, -1.0, -1.0,
							1.0,  1.0, -1.0,
							-1.0,  1.0, -1.0,
							-1.0, -1.0,  1.0,
							1.0, -1.0,  1.0,
							1.0,  1.0,  1.0,
							-1.0,  1.0,  1.0};

	  for (int j = 0; j < 8; j++)
	    for (int i = 0; i < 3; i++)
			coords[j * 3 + i] = coeff[j * 3 + i] * 0.5773502692;

      hipMemcpyToSymbol(HIP_SYMBOL(parCoords),coords.data(),24*sizeof(double));

	  //move surface parametric coordinates to local memory
	  vector<vector<double>> coeff2D{ { -1.0 / sqrt(3.0), -1.0 / sqrt(3.0) },
	  { -1.0 / sqrt(3.0),  1.0 / sqrt(3.0) },
	  { 1.0 / sqrt(3.0), -1.0 / sqrt(3.0) },
	  { 1.0 / sqrt(3.0),  1.0 / sqrt(3.0) } };

	  vector<double> coordsSurf(72);

	  for (int ip = 0; ip < 4; ++ip) {
		  coordsSurf[ip * 18] = coeff2D[ip][0];
		  coordsSurf[ip * 18 + 1] = coeff2D[ip][1];
		  coordsSurf[ip * 18 + 2] = 1.0;

		  coordsSurf[ip * 18 + 3] = coeff2D[ip][0];
		  coordsSurf[ip * 18 + 4] = coeff2D[ip][1];
		  coordsSurf[ip * 18 + 5] = -1.0;

		  coordsSurf[ip * 18 + 6] = coeff2D[ip][0];
		  coordsSurf[ip * 18 + 7] = -1.0;
		  coordsSurf[ip * 18 + 8] = coeff2D[ip][1];

		  coordsSurf[ip * 18 + 9] = coeff2D[ip][0];
		  coordsSurf[ip * 18 + 10] = 1.0;
		  coordsSurf[ip * 18 + 11] = coeff2D[ip][1];

		  coordsSurf[ip * 18 + 12] = -1.0;
		  coordsSurf[ip * 18 + 13] = coeff2D[ip][0];
		  coordsSurf[ip * 18 + 14] = coeff2D[ip][1];

		  coordsSurf[ip * 18 + 15] = 1.0;
		  coordsSurf[ip * 18 + 16] = coeff2D[ip][0];
		  coordsSurf[ip * 18 + 17] = coeff2D[ip][1];
	  }

	  hipMemcpyToSymbol(HIP_SYMBOL(parCoordsSurf), coordsSurf.data(), 72 * sizeof(double));

	  // 2D surface parametric coordinates to constant memory
	  vector<double> coords2D{ -0.5773502692, -0.5773502692,
		  -0.5773502692,  0.5773502692,
		  0.5773502692, -0.5773502692,
		  0.5773502692,  0.5773502692 };

	  hipMemcpyToSymbol(HIP_SYMBOL(parCoords2D), coords2D.data(), 8 * sizeof(double));

		vector<int> mapIndx{ 0,  1,  2,  3,  4,  5,  6,  7,
							1,  8,  9, 10, 11, 12, 13, 14 ,
							2,  9, 15, 16, 17, 18, 19, 20 ,
							3, 10, 16, 21, 22, 23, 24, 25 ,
							4, 11, 17, 22, 26, 27, 28, 29 ,
							5, 12, 18, 23, 27, 30, 31, 32 ,
							6, 13, 19, 24, 28, 31, 33, 34 ,
							7, 14, 20, 25, 29, 32, 34, 35 };

	  hipMemcpyToSymbol(HIP_SYMBOL(mapIndex), mapIndx.data(), 8 * sizeof(int));

      hipEventRecord(stop, 0);
      hipEventSynchronize(stop);
      float elapsedTime;
      hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
      // Clean up:
      hipEventDestroy(start);
      hipEventDestroy(stop);

	  std::cout << "Device data setup took " << (double)elapsedTime/1000 << " seconds" << std::endl;

}

void AllocateDeviceData(elementData& elem) {
	int nn = elem.nn;
	int numEl = elem.numEl;
	int numSurf = elem.numSurf;

	//Allocate device arrays
	checkCudaErrors(hipMalloc((void**)&elem.dGlobMass, elem.rhsCountEle*nn*sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&elem.dGlobRHS, 12 * nn * sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&elem.dEleStiffness, 36*numEl*sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&elem.dEleNodes, 8*numEl*sizeof(int)));
	checkCudaErrors(hipMalloc((void**)&elem.dEleNodeCoords, 24*numEl*sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&elem.dEleMat, 6*numEl*sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&elem.dEleBirthTimes, numEl*sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&elem.dthetaN, nn*sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&elem.dNUniId, 8*numEl * sizeof(int)));

	checkCudaErrors(hipMalloc((void**)&elem.dSurfNodes, 4 * numSurf * sizeof(int)));
	checkCudaErrors(hipMalloc((void**)&elem.dSurfNodeCoords, 8 * numSurf * sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&elem.dSurfIndx, 4 * numSurf * sizeof(int)));
	checkCudaErrors(hipMalloc((void**)&elem.dSurfPlane, numSurf * sizeof(int)));
	checkCudaErrors(hipMalloc((void**)&elem.dSurfFlux, 5 * numSurf * sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&elem.dSurfBirthElem, numSurf * sizeof(int)));
	checkCudaErrors(hipMalloc((void**)&elem.dGlobRHS_Surf, nn*elem.rhsCount * sizeof(double)));

	checkCudaErrors(hipMalloc((void**)&elem.dFixedNodes, elem.numFixed * sizeof(int)));
	checkCudaErrors(hipMalloc((void**)&elem.dFixedNodeVals, elem.numFixed * sizeof(double)));

	checkCudaErrors(hipMalloc((void**)&elem.dBirthNodes, nn * sizeof(int)));


	hipMemset(elem.dEleStiffness, 0, 36*numEl*sizeof(double));
}

void CopyToDevice(elementData& elem) {
	int nn = elem.nn;
	int numEl = elem.numEl;
	int numSurf = elem.numSurf;
	checkCudaErrors(hipMemcpy(elem.dEleNodes,elem.eleNodes.data(),8*numEl*sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(elem.dEleNodeCoords,elem.eleNodeCoords.data(),24*numEl*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(elem.dEleMat,elem.eleMat.data(), 6*numEl*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(elem.dEleBirthTimes,elem.eleBirthTimes.data(),numEl*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(elem.dthetaN, elem.thetaN.data(), nn*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(elem.dNUniId, elem.nUniId.data(), 8*numEl * sizeof(int), hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(elem.dSurfNodes, elem.surfNodes.data(), 4 * numSurf * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(elem.dSurfIndx, elem.surfIndx.data(), 4 * numSurf * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(elem.dSurfNodeCoords, elem.surfNodeCoords.data(), 8 * numSurf * sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(elem.dSurfPlane, elem.surfPlane.data(), numSurf * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(elem.dSurfFlux, elem.surfFlux.data(), 5 * numSurf * sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(elem.dSurfBirthElem, elem.surfBirthElem.data(), numSurf * sizeof(int), hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(elem.dFixedNodes, elem.fixedNodes, elem.numFixed * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(elem.dFixedNodeVals, elem.fixedValues, elem.numFixed * sizeof(double), hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(elem.dBirthNodes, elem.birthNodes.data(), nn * sizeof(int), hipMemcpyHostToDevice));
}

void CopyToHost(elementData& elem) {
	int nn = elem.nn;
	//int numEl = elem.numEl;

	//checkCudaErrors(hipMemcpy(elem.eleStiffness.data(), elem.dEleStiffness, 36*numEl*sizeof(double), hipMemcpyDeviceToHost));
	//checkCudaErrors(hipMemcpy(elem.globMass.data(), elem.dGlobMass, 12*nn*sizeof(double), hipMemcpyDeviceToHost));
	//checkCudaErrors(hipMemcpy(elem.globRHS.data(), elem.dGlobRHS, 12*nn * sizeof(double), hipMemcpyDeviceToHost));
	//checkCudaErrors(hipMemcpy(elem.globRHS_Surf.data(), elem.dGlobRHS_Surf, elem.rhsCount * nn * sizeof(double), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(elem.thetaN.data(), elem.dthetaN,  nn * sizeof(double), hipMemcpyDeviceToHost));
}

void FreeDevice(elementData& elem) {
	checkCudaErrors(hipFree(elem.dGlobMass));
	checkCudaErrors(hipFree(elem.dEleStiffness));
	checkCudaErrors(hipFree(elem.dEleNodes));
	checkCudaErrors(hipFree(elem.dEleNodeCoords));
	checkCudaErrors(hipFree(elem.dEleMat));
	checkCudaErrors(hipFree(elem.dEleBirthTimes));
	checkCudaErrors(hipFree(elem.dthetaN));
	checkCudaErrors(hipFree(elem.dNUniId));
	checkCudaErrors(hipFree(elem.dGlobRHS));

	checkCudaErrors(hipFree(elem.dSurfNodes));
	checkCudaErrors(hipFree(elem.dSurfIndx));
	checkCudaErrors(hipFree(elem.dSurfNodeCoords));
	checkCudaErrors(hipFree(elem.dSurfPlane));
	checkCudaErrors(hipFree(elem.dSurfFlux));
	checkCudaErrors(hipFree(elem.dSurfBirthElem));
	checkCudaErrors(hipFree(elem.dGlobRHS_Surf));

	checkCudaErrors(hipFree(elem.dFixedNodes));
	checkCudaErrors(hipFree(elem.dFixedNodeVals));

	checkCudaErrors(hipFree(elem.dBirthNodes));
}

void clearDeviceData(elementData& elem) {
	//Clear some arrays
	int nn = elem.nn;
	hipMemset(elem.dGlobMass, 0, 12*nn*sizeof(double));
	hipMemset(elem.dGlobRHS, 0, 12 * nn * sizeof(double));
	hipMemset(elem.dGlobRHS_Surf, 0, elem.rhsCount*nn * sizeof(double));
}

void initializeStiffnessOnD(elementData& elemData) {
	hipError_t cudaStatus;
	int gridSize = elemData.numEl/256 + 1;
	int blockSize = 256;

    //Start recording time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Start record
    hipEventRecord(start, 0);

	initializeStiffness<<<gridSize, blockSize>>>(elemData.dEleMat, elemData.dEleNodeCoords, elemData.dEleStiffness,
				elemData.dEleNodes, elemData.numEl, elemData.nn);

	cudaStatus = hipGetLastError();
	if(cudaStatus != hipSuccess) {
		std::cout <<"initialiazeStiffness Kernel failed: "<<hipGetErrorString(cudaStatus)<< std::endl;
		FreeDevice(elemData);
	}

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
    // Clean up:
    hipEventDestroy(start);
    hipEventDestroy(stop);

	//std::cout << "initialize Stiffness took " << (double)elapsedTime/1000 << " seconds" << std::endl;
}

void updateMassOnD(elementData& elemData, DomainManager*& domainMgr) {
	hipError_t cudaStatus;


	//find birth point
	auto up = std::upper_bound(domainMgr->elementList_.begin(), domainMgr->elementList_.end(),
		domainMgr->currTime_, [](const double bTime, Element* a) {return a->birthTime_ > bTime; });

	int birthElemPos = up - domainMgr->elementList_.begin();

	int gridSize = birthElemPos / 256 + 1;
	int blockSize = 256;

    //Start recording time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Start record
    hipEventRecord(start, 0);

	updateMass<<<gridSize, blockSize>>>(elemData.dthetaN, elemData.dEleMat, elemData.dEleNodeCoords,
			elemData.dGlobMass, elemData.dEleNodes, elemData.dNUniId, elemData.numEl, elemData.nn, birthElemPos);

	cudaStatus = hipGetLastError();
	if(cudaStatus != hipSuccess) {
		std::cout <<"updateMass Kernel failed: "<<hipGetErrorString(cudaStatus)<< std::endl;
		FreeDevice(elemData);
	}

	gridSize= elemData.nn / 256 + 1;
	massReduce <<<gridSize, blockSize >>> (elemData.dGlobMass, elemData.nn, elemData.rhsCountEle);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		cout << "massReduce Kernel failed: " << hipGetErrorString(cudaStatus) << endl;
		FreeDevice(elemData);
	}


    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
    // Clean up:
    hipEventDestroy(start);
    hipEventDestroy(stop);

	//std::cout << "update Mass took " << (double)elapsedTime/1000 << " seconds" << std::endl;

}

void updateIntForceOnD(elementData& elemData, DomainManager*& domainMgr) {
	hipError_t cudaStatus;

	//find birth point
	auto up = std::upper_bound(domainMgr->elementList_.begin(), domainMgr->elementList_.end(),
		domainMgr->currTime_, [](const double bTime, Element* a) {return a->birthTime_ > bTime; });

	int birthElemPos = up - domainMgr->elementList_.begin();

	int gridSize = birthElemPos / 256 + 1;
	int blockSize = 256;

	//Start recording time
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Start record
	hipEventRecord(start, 0);

	getInternalForce <<<gridSize, blockSize >>> (elemData.dGlobRHS, elemData.dEleStiffness, elemData.dthetaN,
		elemData.dEleNodes, elemData.dNUniId, elemData.numEl, elemData.nn, birthElemPos);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		std::cout << "Internal-force Kernel failed: " << hipGetErrorString(cudaStatus) << std::endl;
		FreeDevice(elemData);
	}

	gridSize = elemData.nn / 256 + 1;
	interForceReduce <<<gridSize, blockSize >>> (elemData.dGlobRHS, elemData.nn);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		cout << "interForceReduce Kernel failed: " << hipGetErrorString(cudaStatus) << endl;
		FreeDevice(elemData);
	}


	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
													 // Clean up:
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//std::cout << "Internal force took " << (double)elapsedTime / 1000 << " seconds" << std::endl;

}

void updateFluxKernel(elementData& elemData, DomainManager*& domainMgr) {
	hipError_t cudaStatus;

	//Start recording time
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Start record
	hipEventRecord(start, 0);

	/*
	*
	* Tool position calculation
	*
	*/

	// current point on toolpath
	auto up = std::upper_bound(domainMgr->tooltxyz_.begin(), domainMgr->tooltxyz_.end(), domainMgr->currTime_,
		[](const double Ctime, vector<double, allocator<double>> a) {return a[0] > Ctime; });

	if(up == domainMgr->tooltxyz_.end())
		up--;

	int toolpathIndex = up - domainMgr->tooltxyz_.begin();

	vector<double> & txyzN = domainMgr->tooltxyz_[toolpathIndex];
	vector<double> & txyzNminus = domainMgr->tooltxyz_[toolpathIndex - 1];

	double num = domainMgr->currTime_ - txyzNminus[0];
	double den = txyzN[0] - txyzNminus[0];
	double rat = num / den;
	elemData.tool0 = rat * (txyzN[1] - txyzNminus[1]) + txyzNminus[1];
	elemData.tool1 = rat * (txyzN[2] - txyzNminus[2]) + txyzNminus[2];
	elemData.tool2 = rat * (txyzN[3] - txyzNminus[3]) + txyzNminus[3];
	elemData.laserState = domainMgr->laserOn_[toolpathIndex];
	// ---------------------------------------- END OF TOOL CALCULATION ------------------------------//


	//find birth point
	auto up2 = std::upper_bound(elemData.boundSurfBirthTime.begin(), elemData.boundSurfBirthTime.end(),
		domainMgr->currTime_, [](double a, double bTime) {return a < bTime; });

	int birthSurfPos = up2 - elemData.boundSurfBirthTime.begin();



	int gridSize = birthSurfPos / 256 + 1;
	int blockSize = 256;


	applyFlux <<<gridSize, blockSize >>>(elemData.dSurfFlux, elemData.dSurfPlane, elemData.dthetaN, elemData.dSurfIndx,elemData.dSurfNodeCoords, elemData.dEleNodeCoords,
		elemData.dSurfNodes, elemData.dSurfBirthElem, elemData.dGlobRHS_Surf, elemData.numSurf, elemData.ambient, elemData.abszero, elemData.tool0, elemData.tool1, elemData.tool2,
		elemData.laserState, birthSurfPos, elemData.sigma, elemData.numEl, elemData.nn);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		std::cout << "Apply Flux Kernel failed: " << hipGetErrorString(cudaStatus) << std::endl;
		FreeDevice(elemData);
	}

	gridSize = elemData.nn / 256 + 1;
	massFlux <<<gridSize, blockSize >>>(elemData.dGlobRHS_Surf, elemData.nn, elemData.rhsCount, elemData.dGlobRHS);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		std::cout << "Mass Flux Kernel failed: " << hipGetErrorString(cudaStatus) << std::endl;
		FreeDevice(elemData);
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
													 // Clean up:
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//std::cout << "Apply Flux Kernel took " << (double)elapsedTime / 1000 << " seconds" << std::endl;


}

void advanceTimeKernel(elementData& elemData, DomainManager*& domainMgr) {
	hipError_t cudaStatus;

	//Start recording time
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Start record
	hipEventRecord(start, 0);

	//get active node count
	auto up = std::upper_bound(elemData.birthNodeTimes.begin(), elemData.birthNodeTimes.end(), domainMgr->currTime_,
			[](double a, double bTime) {return a < bTime; });

	int nodeCount = up - elemData.birthNodeTimes.begin();


	int gridSize = nodeCount / 256 + 1;
	int blockSize = 256;

	advanceTime <<<gridSize, blockSize>>>(elemData.dthetaN, elemData.dGlobMass, elemData.dGlobRHS, elemData.dBirthNodes ,elemData.dt,nodeCount);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		std::cout << "Apply Flux Kernel failed: " << hipGetErrorString(cudaStatus) << std::endl;
		FreeDevice(elemData);
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
													 // Clean up:
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//std::cout << "Advance Time Kernel took " << (double)elapsedTime / 1000 << " seconds" << std::endl;

}

void dirichletBCKernel(elementData& elemData) {
	hipError_t cudaStatus;

	//Start recording time
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Start record
	hipEventRecord(start, 0);

	int gridSize = elemData.numFixed / 256 + 1;
	int blockSize = 256;

	prescribeDirichlet <<<gridSize, blockSize>>> (elemData.dthetaN, elemData.dFixedNodes, elemData.dFixedNodeVals, elemData.numFixed);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		std::cout << "Apply Flux Kernel failed: " << hipGetErrorString(cudaStatus) << std::endl;
		FreeDevice(elemData);
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
													 // Clean up:
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//std::cout << "Prescribe Dirichlet  Kernel took " << (double)elapsedTime / 1000 << " seconds" << std::endl;
}

__global__ void updateMass(const double* __restrict__ thetaN, const double* __restrict__ eleMat, const double* __restrict__ eleNodeCoords,
		double* globMass, const int* __restrict__ eleNodes, const int* __restrict__ nUniId, const int numEl, const int nn, const int numElAct) {

	int idx = blockIdx.x*blockDim.x + threadIdx.x;

	__shared__ int parCoordsS[24];

	if(idx < 24)
		parCoordsS[idx] = parCoords[idx];

	__syncthreads();

	if(idx < numElAct) {

		  // elemental material properties
		double liquidus = eleMat[idx + 2*numEl];
		double solidus = eleMat[idx + numEl];
		double latent = eleMat[idx + 3*numEl];
		double cp =  eleMat[idx + 4*numEl];
		double rho = eleMat[idx];

		double coords[24];

		//shape function
		double N[8];
		double chsi;
		double eta;
		double zeta;

		double deriv[24];
		double Jac[9];

		double thetaIp = 0.0;
		double cpPoint = 0;
		double detJac = 0;

		for(int ip = 0; ip < 8; ++ip) {

			//compute shape function
			chsi = parCoordsS[ip*3 + 0];
			eta = parCoordsS[ip*3 + 1];
			zeta = parCoordsS[ip*3 + 2];
			N[0] = 0.125*(1.0 - chsi)*(1.0 - eta)*(1.0 - zeta);
			N[3] = 0.125*(1.0 - chsi)*(1.0 + eta)*(1.0 - zeta);
			N[2] = 0.125*(1.0 + chsi)*(1.0 + eta)*(1.0 - zeta);
			N[1] = 0.125*(1.0 + chsi)*(1.0 - eta)*(1.0 - zeta);
			N[4] = 0.125*(1.0 - chsi)*(1.0 - eta)*(1.0 + zeta);
			N[7] = 0.125*(1.0 - chsi)*(1.0 + eta)*(1.0 + zeta);
			N[6] = 0.125*(1.0 + chsi)*(1.0 + eta)*(1.0 + zeta);
			N[5] = 0.125*(1.0 + chsi)*(1.0 - eta)*(1.0 + zeta);

			//Calculate temperature at integration points
			for(int i = 0; i < 8; ++i) {
				int ig = eleNodes[numEl*i + idx];
				thetaIp += N[i]*thetaN[ig];
			}

			//compute cp
			cpPoint = (thetaIp <= solidus && thetaIp >= liquidus) ? (cp + latent / ( liquidus - solidus )) : cp;

			//compute derivative of shape functions
			// with respect to chsi
			deriv[0] = -0.1250 * (1 - eta) * (1 - zeta);
			deriv[2] =  0.1250 * (1 + eta) * (1 - zeta);
			deriv[4] = -0.1250 * (1 - eta) * (1 + zeta);
			deriv[6] =  0.1250 * (1 + eta) * (1 + zeta);
			for (int i = 0; i < 4; i++)
				deriv[i * 2 + 1] = -deriv[i * 2];

			// with respect to eta
			deriv[0 + 8] = -0.1250 * (1 - chsi) * (1 - zeta);
			deriv[3 + 8] = -deriv[8];
			deriv[1 + 8] = -0.1250 * (1 + chsi) * (1 - zeta);
			deriv[2 + 8] = -deriv[9];
			deriv[4 + 8] = -0.1250 * (1 - chsi) * (1 + zeta);
			deriv[7 + 8] = -deriv[12];
			deriv[5 + 8] = -0.1250 * (1 + chsi) * (1 + zeta);
			deriv[6 + 8] = -deriv[13];

			// with respect to zeta
			deriv[4 + 16] = 0.1250 * (1 - chsi) * (1 - eta);
			deriv[5 + 16] = 0.1250 * (1 + chsi) * (1 - eta);
			deriv[6 + 16] = 0.1250 * (1 + chsi) * (1 + eta);
			deriv[7 + 16] = 0.1250 * (1 - chsi) * (1 + eta);
			for (int i = 0; i < 4; i++)
				deriv[i + 16] = -deriv[i + 20];

			// get coordinates
			for(int i = 0; i < 8; i++) {
				coords[3*i] = eleNodeCoords[3*numEl*i  + idx];
				coords[3*i + 1] = eleNodeCoords[3*numEl*i + numEl + idx];
				coords[3*i + 2] = eleNodeCoords[3*numEl*i + 2*numEl + idx];
			 }

			for (int k = 0; k < 9; k++)
				Jac[k] = 0;

			// Compute Jacobian
			for (int k = 0; k < 3; k++)
				for (int j = 0; j < 3; j++)
					for (int i = 0; i < 8; i++)
						Jac[k* 3 + j] += deriv[k * 8 + i] * coords[i * 3 + j];


			//determinant of Jacobian
			detJac = Jac[0] * Jac[4] * Jac[8] + Jac[1] * Jac[5] * Jac[6] +
					Jac[3] * Jac[7] * Jac[2] - Jac[2] * Jac[4] * Jac[6] -
					Jac[0] * Jac[5] * Jac[7] - Jac[1] * Jac[3] * Jac[8];

			//Calculate mass matrix
			for (int i = 0; i < 8; i++) {
				int ig = eleNodes[numEl*i + idx];
				for (int j = 0; j < 8; j++)
					globMass[nUniId[i*numEl+idx]*nn + ig] += N[i] * rho * cpPoint * N[j] * detJac;
			}

		}
	}

}

__global__ void initializeStiffness(const double* __restrict__ eleMat, const double* __restrict__ eleNodeCoords,
		double* eleStiffness, const int* __restrict__ eleNodes, const int numEl, const int nn) {

	int idx = blockIdx.x*blockDim.x + threadIdx.x;

	__shared__ int parCoordsS[24];

	if(idx < 24)
		parCoordsS[idx] = parCoords[idx];

	__syncthreads();

	if(idx < numEl) {

		double coords[24];

		//shape function

		double chsi;
		double eta;
		double zeta;

		double iJac[9];
		double Jac[9];
		double gradN[24];
		double deriv[24];
		double swap;


		double detJac = 0;

		double kappa = eleMat[5*numEl + idx];

		for(int ip = 0; ip < 8; ++ip) {

			for(int i = 0; i < 24; i++)
				gradN[i] = 0;

			for(int i = 0; i < 9; i++)
				Jac[i] = 0;

			//compute shape function
			chsi = parCoordsS[ip*3 + 0];
			eta = parCoordsS[ip*3 + 1];
			zeta = parCoordsS[ip*3 + 2];

			//compute derivative of shape functions

			// with respect to chsi
			deriv[0] = -0.1250 * (1 - eta) * (1 - zeta);
			deriv[2] =  0.1250 * (1 + eta) * (1 - zeta);
			deriv[4] = -0.1250 * (1 - eta) * (1 + zeta);
			deriv[6] =  0.1250 * (1 + eta) * (1 + zeta);
			for (int i = 0; i < 4; i++)
				deriv[i * 2 + 1] = -deriv[i * 2];

			// with respect to eta
			deriv[0 + 8] = -0.1250 * (1 - chsi) * (1 - zeta);
			deriv[3 + 8] = -deriv[8];
			deriv[1 + 8] = -0.1250 * (1 + chsi) * (1 - zeta);
			deriv[2 + 8] = -deriv[9];
			deriv[4 + 8] = -0.1250 * (1 - chsi) * (1 + zeta);
			deriv[7 + 8] = -deriv[12];
			deriv[5 + 8] = -0.1250 * (1 + chsi) * (1 + zeta);
			deriv[6 + 8] = -deriv[13];

			// with respect to zeta
			deriv[4 + 16] = 0.1250 * (1 - chsi) * (1 - eta);
			deriv[5 + 16] = 0.1250 * (1 + chsi) * (1 - eta);
			deriv[6 + 16] = 0.1250 * (1 + chsi) * (1 + eta);
			deriv[7 + 16] = 0.1250 * (1 - chsi) * (1 + eta);
			for (int i = 0; i < 4; i++)
				deriv[i + 16] = -deriv[i + 20];

			// get coordinates
			for(int i = 0; i < 8; i++) {
				coords[3*i] = eleNodeCoords[3*numEl*i  + idx];
				coords[3*i + 1] = eleNodeCoords[3*numEl*i + numEl + idx];
				coords[3*i + 2] = eleNodeCoords[3*numEl*i + 2*numEl + idx];
			 }


			// Compute Jacobian
			for (int k = 0; k < 3; k++)
				for (int j = 0; j < 3; j++)
					for (int i = 0; i < 8; i++)
						Jac[k* 3 + j] += deriv[k * 8 + i] * coords[i * 3 + j];

			//determinant of Jacobian
			detJac = Jac[0] * Jac[4] * Jac[8] + Jac[1] * Jac[5] * Jac[6] +
					Jac[3] * Jac[7] * Jac[2] - Jac[2] * Jac[4] * Jac[6] -
					Jac[0] * Jac[5] * Jac[7] - Jac[1] * Jac[3] * Jac[8];


			//compute inverse of Jacobian

			iJac[0] = (1/detJac) * (Jac[4] * Jac[8] - Jac[5] * Jac[7]);
			iJac[1] = (-1/detJac) * (Jac[3] * Jac[8] - Jac[5] * Jac[6]);
			iJac[2] = (1/detJac) * (Jac[3] * Jac[7] - Jac[4] * Jac[6]);

			iJac[3] = (-1/detJac) * (Jac[1] * Jac[8] - Jac[2] * Jac[7]);
			iJac[4] = (1/detJac) * (Jac[0] * Jac[8] - Jac[2] * Jac[6]);
			iJac[5] = (-1/detJac) * (Jac[0] * Jac[7] - Jac[1] * Jac[6]);

			iJac[6] = (1/detJac) * (Jac[1] * Jac[5] - Jac[2] * Jac[4]);
			iJac[7] = (-1/detJac) * (Jac[0] * Jac[5] - Jac[2] * Jac[3]);
			iJac[8] = (1/detJac) * (Jac[0] * Jac[4] - Jac[1] * Jac[3]);

			swap = iJac[1];
			iJac[1] = iJac[3];
			iJac[3] = swap;

			swap = iJac[2];
			iJac[2] = iJac[6];
			iJac[6] = swap;

			swap = iJac[5];
			iJac[5] = iJac[7];
			iJac[7] = swap;

			//compute derivative of shape function w.r.t real coordinates


			for (int k = 0; k < 8; k++)
				for (int j = 0; j < 3; j++)
				  for (int i = 0; i < 3; i++)
					gradN[j*8 + k] += iJac[j*3 + i] * deriv[i*8 + k];


			//Calculate element stiffness matrix
			int count = 0;
			for (int i=0; i<8; i++)
			  for (int j = i; j < 8; j++) {
				for (int k = 0; k < 3; k++)
					eleStiffness[count*numEl + idx] += gradN[k * 8 + i] * kappa * gradN[k * 8 + j] * detJac;

				count++;
			  }
		}
	}
}

__global__ void massReduce(double* __restrict__ globMass, const int nn, const int rhsCount) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < nn)
		for (int elInd = 1; elInd < rhsCount; elInd++)
			globMass[idx] += globMass[idx + nn*elInd];

}

__global__ void getInternalForce(double* globRHS, double* __restrict__ eleStiffness, double* __restrict__ thetaN, 
	const int* __restrict__ eleNodes, const int* __restrict__ nUniId, const int numEl, const int nn, const int numElAct) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;

	__shared__ int mapIndexS[64];

	if(idx < 64)
		mapIndexS[idx] = mapIndex[idx];

	__syncthreads();

	if(idx < numElAct) {
		for (int row = 0; row < 8; row++)
		{
			int ig = eleNodes[numEl*row + idx];
			for (int col = 0; col < 8; col++)
			{
				int stiffInd = mapIndexS[row*8 + col];
				globRHS[nUniId[row*numEl + idx] * nn + ig] -= eleStiffness[idx + stiffInd*numEl] * thetaN[eleNodes[idx + col*numEl]];
			}
		}
	}
}

__global__ void interForceReduce(double* __restrict__ globRHS, const int nn) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < nn)
		for (int elInd = 1; elInd < 12; elInd++)
			globRHS[idx] += globRHS[idx + nn*elInd];
}

__global__ void applyFlux(const double* __restrict__ surfFlux, const int* __restrict__ surfPlane, const double* __restrict__ thetaN, const int* __restrict__ surfIndx,
	const double* __restrict__ surfNodeCoords, const double* __restrict__ eleNodeCoords, const int* __restrict__ surfNodes, const int* __restrict__ surfElemBirth,
	double* rhs, int numSurf, double ambient, double abszero, double tool0, double tool1, double tool2, int laserState, int numSurfAct, double sigma, int numEl, int nn) {

	int idx = blockIdx.x*blockDim.x + threadIdx.x;

	//move to shared memory for speed
	__shared__ int parCoordsSurfS[72];
	__shared__ int parCoords2DS[8];

	if(idx < 8)
		parCoords2DS[idx] = parCoords2D[idx];

	if(idx < 72)
		parCoordsSurfS[idx] = parCoordsSurf[idx];

	__syncthreads();

	if (idx < numSurfAct) {
		//shape function
		double N[8];
		double chsi;
		double eta;
		double zeta;
		int plane;

		//global location of integration points
		double xip;
		double yip;
		double zip;
		double r2;

		//Convection, Radiation, and Moving Flux
		double thetaIp;
		double qconv;
		double ambient4;
		double thetaIp4;
		double qrad;
		double qmov;


		const double conv1 = surfFlux[idx];
		const double conv2 = surfFlux[idx + numSurf];
		const double rad1 = surfFlux[idx + 2 * numSurf];
		const double rad2 = surfFlux[idx + 3 * numSurf];
		const double heat = surfFlux[idx + 4 * numSurf];
		const double rb2 = 1.21;
		const double Qin = 892.5;
		const double pi = 3.14159265358979;

		double Jac[4];
		int element;
		double coords[8];
		double detJac;


		for (int ip = 0; ip < 4; ++ip) {

			//compute shape function for surface

			//get surface plane
			plane = surfPlane[idx];

			//integration points
			chsi = parCoordsSurfS[ip * 18 + plane * 3];
			eta = parCoordsSurfS[ip * 18 + plane * 3 + 1];
			zeta = parCoordsSurfS[ip * 18 + plane * 3 + 2];

			//shape function in parametric coordinates
			N[0] = 0.125*(1.0 - chsi)*(1.0 - eta)*(1.0 - zeta);
			N[3] = 0.125*(1.0 - chsi)*(1.0 + eta)*(1.0 - zeta);
			N[2] = 0.125*(1.0 + chsi)*(1.0 + eta)*(1.0 - zeta);
			N[1] = 0.125*(1.0 + chsi)*(1.0 - eta)*(1.0 - zeta);
			N[4] = 0.125*(1.0 - chsi)*(1.0 - eta)*(1.0 + zeta);
			N[7] = 0.125*(1.0 - chsi)*(1.0 + eta)*(1.0 + zeta);
			N[6] = 0.125*(1.0 + chsi)*(1.0 + eta)*(1.0 + zeta);
			N[5] = 0.125*(1.0 + chsi)*(1.0 - eta)*(1.0 + zeta);

			xip = 0;
			yip = 0;
			zip = 0;

			//global location of integration points
			element = surfElemBirth[idx];
			for (int i = 0; i < 8; ++i) {
				xip += N[i] * eleNodeCoords[3 * numEl*i + element];
				yip += N[i] * eleNodeCoords[3 * numEl*i + numEl + element];
				zip += N[i] * eleNodeCoords[3 * numEl*i + 2 * numEl + element];

				coords[i] = 0;
			}

			//get mapped coordinates,

			//reuse Nodal variable to store GradN
			//2D shape functions
			chsi = parCoords2DS[ip * 2];
			eta = parCoords2DS[ip * 2 + 1];

			N[0] = 0.25 * (eta - 1.0);   /// remember this is gradN not N, need to save up on registers
			N[1] = 0.25 * (1.0 - eta);
			N[2] = 0.25 * (1.0 + eta);
			N[3] = -0.25 * (1.0 + eta);
			N[4] = 0.25 * (chsi - 1.0);
			N[5] = -0.25 * (1.0 + chsi);
			N[6] = 0.25 * (1.0 + chsi);
			N[7] = 0.25 * (1.0 - chsi);

			//Calculate Jacobian
			for (int i = 0; i < 4; ++i)
				Jac[i] = 0.0;

			for (int i = 0; i < 4; ++i) {
				coords[i * 2] = surfNodeCoords[2 * numSurf*i + idx];
				coords[i * 2 + 1] = surfNodeCoords[2 * numSurf*i + numSurf + idx];
			}

			for (int i = 0; i < 2; i++)
				for (int j = 0; j < 4; j++)
					for (int k = 0; k < 2; k++)
						Jac[i * 2 + k] += N[i * 4 + j] * coords[j * 2 + k];

			detJac = Jac[0] * Jac[3] - Jac[2] * Jac[1];

			// Use N again to calculate the 2D shape function for rhs

			N[0] = 0.25 * (1 - chsi) * (1 - eta);
			N[1] = 0.25 * (1 + chsi) * (1 - eta);
			N[2] = 0.25 * (1 + chsi) * (1 + eta);
			N[3] = 0.25 * (1 - chsi) * (1 + eta);

			thetaIp = 0.0;
			//Calculate
			for (int i = 0; i < 4; i++) {
				int ig = surfNodes[numSurf*i + idx];
				thetaIp += N[i] * thetaN[ig];
			}

			//convection
			qconv = -conv2 * (thetaIp - ambient);

			//radiation
			ambient4 = (ambient - abszero)*(ambient - abszero)*(ambient - abszero)*(ambient - abszero);
			thetaIp4 = (thetaIp - abszero)*(thetaIp - abszero)*(thetaIp - abszero)*(thetaIp - abszero);

			qrad = -sigma * rad2 * (thetaIp4 - ambient4);


			//moving flux from laser
			r2 = ((xip - tool0) * (xip - tool0) +
				(yip - tool1) * (yip - tool1) +
				(zip - tool2) * (zip - tool2));

			const double val = 3.0 * Qin / (pi * rb2) * exp(-3.0 * r2 / rb2);

			qmov = (laserState == 1) ? val : 0.0;

			for (int i = 0; i < 4; ++i) {
				int ig = surfNodes[numSurf*i + idx];
				int ir = surfIndx[idx*4 + i];
				rhs[ig + ir*nn] += N[i] * detJac * (conv1*qconv + rad1*qrad + heat*qmov);
			}
		}
	}
}

__global__ void massFlux(double* rhs, const int nn, int rhsCount, double* globalRhs) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < nn)
		for (int elInd = 0; elInd < rhsCount; elInd++)
			globalRhs[idx] += rhs[idx + nn*elInd];
}

__global__ void advanceTime(double* thetaN, const double* __restrict__ globalMass, const double* __restrict__ rhs, const int* __restrict__ birthNodes,double dt, int numActNodes) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;

	if(idx < numActNodes) {
		int ig = birthNodes[idx];
		thetaN[ig] += dt * (rhs[ig])/globalMass[ig];
	}
}

__global__ void prescribeDirichlet(double* thetaN, const int* __restrict__ fixedNodes, const double* __restrict__ fixedNodeVals, int numFixed) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;

	if(idx < numFixed) {
		int node = fixedNodes[idx];
		thetaN[node] = fixedNodeVals[idx];
	}
}
