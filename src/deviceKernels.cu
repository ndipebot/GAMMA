#include "hip/hip_runtime.h"

/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <deviceHeader.h>
#include <iostream>
#include <hip/hip_runtime_api.h>

__constant__ double parCoords[24];

__global__ void updateMass(const double* __restrict__ thetaN, const double* __restrict__ eleMat, const double* __restrict__ eleNodeCoords,
		double* globMass, const int* __restrict__ eleNodes, const int numEl, const int nn);

__global__ void initializeStiffness(const double* __restrict__ eleMat, const double* __restrict__ eleNodeCoords, double* eleStiffness,
		const int* __restrict__ eleNodes, const int numEl, const int nn);

void createDataOnDeveice(DomainManager*& domainMgr, elementData& elemData, double initTemp) {
    //Start recording time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Start record
    hipEventRecord(start, 0);


	int numEl = domainMgr->nel_;
	int nn = domainMgr->nn_;


	elemData.initTemp = initTemp;
	elemData.nn = nn;
	elemData.numEl = numEl;

	//Element data
	elemData.eleNodes.resize(8*numEl);
	elemData.eleNodeCoords.resize(24*numEl);
	elemData.eleMat.resize(6*numEl);
	elemData.eleBirthTimes.resize(numEl);
	elemData.thetaN.resize(nn);

	elemData.eleStiffness.resize(numEl*36);
	elemData.globMass.resize(nn*8);

	// set element values
	for(const auto & elem : domainMgr->elementList_) {
		 int * nodes = elem->nID_;
		 int eID = &elem - &domainMgr->elementList_[0]; // grab current element id


		 for(int i = 0; i < 8; ++i) {
			 //assign nodes to device vector in coalesced manner
			 elemData.eleNodes[numEl*i + eID] = nodes[i];
			 //grab coalesced nodal coordinates while you're at it
			 for(int j = 0; j < 3; ++j)
				 elemData.eleNodeCoords[3*numEl*i + j*numEl + eID] = domainMgr->coordList_[nodes[i]*3+j];
		 }
		//material properties
		elemData.eleMat[eID] = elem->rho_;
		elemData.eleMat[numEl + eID] = elem->solidus_;
		elemData.eleMat[2*numEl + eID] = elem->liquidus_;
		elemData.eleMat[3*numEl + eID] = elem->latent_;
		elemData.eleMat[4*numEl + eID] = elem->cp_;
		elemData.eleMat[5*numEl + eID] = elem->cond_;

	    //element birth times
		elemData.eleBirthTimes[eID] = elem->birthTime_;
	}

	AllocateDeviceData(elemData);
	CopyToDevice(elemData);

	//move parametric coordinates to local memory

	vector<double> coords(24);

	vector<double> coeff {-1.0, -1.0, -1.0,
							1.0, -1.0, -1.0,
							1.0,  1.0, -1.0,
							-1.0,  1.0, -1.0,
							-1.0, -1.0,  1.0,
							1.0, -1.0,  1.0,
							1.0,  1.0,  1.0,
							-1.0,  1.0,  1.0};

	  for (int j = 0; j < 8; j++)
	    for (int i = 0; i < 3; i++)
	      coords[j*3 + i] = coeff[j*3 + i] * 0.5773502692;

      hipMemcpyToSymbol(HIP_SYMBOL(parCoords),coords.data(),24*sizeof(double));

      hipEventRecord(stop, 0);
      hipEventSynchronize(stop);
      float elapsedTime;
      hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
      // Clean up:
      hipEventDestroy(start);
      hipEventDestroy(stop);

      cout << "Device data setup took " << (double)elapsedTime/1000 << " seconds" << endl;

}

void AllocateDeviceData(elementData& elem) {
	int nn = elem.nn;
	int numEl = elem.numEl;

	//Allocate device arrays
	checkCudaErrors(hipMalloc((void**)&elem.dGlobMass, 8*nn*sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&elem.dEleStiffness, 36*numEl*sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&elem.dEleNodes, 8*numEl*sizeof(int)));
	checkCudaErrors(hipMalloc((void**)&elem.dEleNodeCoords, 24*numEl*sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&elem.dEleMat, 6*numEl*sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&elem.dEleBirthTimes, numEl*sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&elem.dthetaN, nn*sizeof(double)));

	//Clear some arrays
	checkCudaErrors(hipMemset(elem.dGlobMass, 0, 8*nn*sizeof(double)));
	checkCudaErrors(hipMemset(elem.dEleStiffness, 0, 36*numEl*sizeof(double)));
	checkCudaErrors(hipMemset(elem.dthetaN, elem.initTemp, nn*sizeof(double)));
}

void CopyToDevice(elementData& elem) {
	int numEl = elem.numEl;

	checkCudaErrors(hipMemcpy(elem.dEleNodes,elem.eleNodes.data(),8*numEl*sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(elem.dEleNodeCoords,elem.eleNodeCoords.data(),24*numEl*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(elem.dEleMat,elem.eleMat.data(), 6*numEl*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(elem.dEleBirthTimes,elem.eleBirthTimes.data(),numEl*sizeof(double), hipMemcpyHostToDevice));
}

void CopyToHost(elementData& elem) {
	int nn = elem.nn;
	int numEl = elem.numEl;

	checkCudaErrors(hipMemcpy(elem.eleStiffness.data(), elem.dEleStiffness, 36*numEl*sizeof(double), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(elem.globMass.data(), elem.dGlobMass, 8*nn*sizeof(double), hipMemcpyDeviceToHost));
}

void FreeDevice(elementData& elem) {
	checkCudaErrors(hipFree(elem.dGlobMass));
	checkCudaErrors(hipFree(elem.dEleStiffness));
	checkCudaErrors(hipFree(elem.dEleNodes));
	checkCudaErrors(hipFree(elem.dEleNodeCoords));
	checkCudaErrors(hipFree(elem.dEleMat));
	checkCudaErrors(hipFree(elem.dEleBirthTimes));
	checkCudaErrors(hipFree(elem.dthetaN));
}


void initializeStiffnessOnD(elementData& elemData) {
	hipError_t cudaStatus;
	int gridSize = elemData.numEl/256 + 1;
	int blockSize = 256;

    //Start recording time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Start record
    hipEventRecord(start, 0);

	initializeStiffness<<<gridSize, blockSize>>>(elemData.dEleMat, elemData.dEleNodeCoords, elemData.dEleStiffness,
				elemData.dEleNodes, elemData.numEl, elemData.nn);

	cudaStatus = hipGetLastError();
	if(cudaStatus != hipSuccess) {
		cout <<"initialiazeStiffness Kernel failed: "<<hipGetErrorString(cudaStatus)<< endl;
		FreeDevice(elemData);
	}

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
    // Clean up:
    hipEventDestroy(start);
    hipEventDestroy(stop);

    cout << "initialize Stiffness took " << (double)elapsedTime/1000 << " seconds" << endl;
}

void updateMassOnD(elementData& elemData) {
	hipError_t cudaStatus;
	int gridSize = elemData.numEl/256 + 1;
	int blockSize = 256;

    //Start recording time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Start record
    hipEventRecord(start, 0);

	updateMass<<<gridSize, blockSize>>>(elemData.dthetaN, elemData.dEleMat, elemData.dEleNodeCoords,
			elemData.dGlobMass, elemData.dEleNodes, elemData.numEl, elemData.nn);

	cudaStatus = hipGetLastError();
	if(cudaStatus != hipSuccess) {
		cout <<"updateMass Kernel failed: "<<hipGetErrorString(cudaStatus)<< endl;
		FreeDevice(elemData);
	}


    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
    // Clean up:
    hipEventDestroy(start);
    hipEventDestroy(stop);

    cout << "update Mass took " << (double)elapsedTime/1000 << " seconds" << endl;

}

__global__ void updateMass(const double* __restrict__ thetaN, const double* __restrict__ eleMat, const double* __restrict__ eleNodeCoords,
		double* globMass, const int* __restrict__ eleNodes, const int numEl, const int nn) {

	int idx = blockIdx.x*blockDim.x + threadIdx.x;

	if(idx < numEl) {

		  // elemental material properties
		double liquidus = eleMat[idx + 2*numEl];
		double solidus = eleMat[idx + numEl];
		double latent = eleMat[idx + 3*numEl];
		double cp =  eleMat[idx + 4*numEl];
		double rho = eleMat[idx];

		double coords[24];

		//shape function
		double N[8];
		double chsi;
		double eta;
		double zeta;

		double deriv[24];
		double Jac[9];

		double thetaIp = 0.0;
		double cpPoint = 0;
		double detJac = 0;

		for(int ip = 0; ip < 8; ++ip) {

			//compute shape function
			chsi = parCoords[ip*3 + 0];
			eta = parCoords[ip*3 + 1];
			zeta = parCoords[ip*3 + 2];
			N[0] = 0.125*(1.0 - chsi)*(1.0 - eta)*(1.0 - zeta);
			N[3] = 0.125*(1.0 - chsi)*(1.0 + eta)*(1.0 - zeta);
			N[2] = 0.125*(1.0 + chsi)*(1.0 + eta)*(1.0 - zeta);
			N[1] = 0.125*(1.0 + chsi)*(1.0 - eta)*(1.0 - zeta);
			N[4] = 0.125*(1.0 - chsi)*(1.0 - eta)*(1.0 + zeta);
			N[7] = 0.125*(1.0 - chsi)*(1.0 + eta)*(1.0 + zeta);
			N[6] = 0.125*(1.0 + chsi)*(1.0 + eta)*(1.0 + zeta);
			N[5] = 0.125*(1.0 + chsi)*(1.0 - eta)*(1.0 + zeta);

			//Calculate temperature at integration points
			for(int i = 0; i < 8; ++i) {
				int ig = eleNodes[numEl*i + idx];
				thetaIp += N[i]*thetaN[ig];
			}

			//compute cp
			cpPoint = (thetaIp <= solidus && thetaIp >= liquidus) ? (cp + latent / ( liquidus - solidus )) : cp;

			//compute derivative of shape functions
			// with respect to chsi
			deriv[0] = -0.1250 * (1 - eta) * (1 - zeta);
			deriv[2] =  0.1250 * (1 + eta) * (1 - zeta);
			deriv[4] = -0.1250 * (1 - eta) * (1 + zeta);
			deriv[6] =  0.1250 * (1 + eta) * (1 + zeta);
			for (int i = 0; i < 4; i++)
				deriv[i * 2 + 1] = -deriv[i * 2];

			// with respect to eta
			deriv[0 + 8] = -0.1250 * (1 - chsi) * (1 - zeta);
			deriv[3 + 8] = -deriv[8];
			deriv[1 + 8] = -0.1250 * (1 + chsi) * (1 - zeta);
			deriv[2 + 8] = -deriv[9];
			deriv[4 + 8] = -0.1250 * (1 - chsi) * (1 + zeta);
			deriv[7 + 8] = -deriv[12];
			deriv[5 + 8] = -0.1250 * (1 + chsi) * (1 + zeta);
			deriv[6 + 8] = -deriv[13];

			// with respect to zeta
			deriv[4 + 16] = 0.1250 * (1 - chsi) * (1 - eta);
			deriv[5 + 16] = 0.1250 * (1 + chsi) * (1 - eta);
			deriv[6 + 16] = 0.1250 * (1 + chsi) * (1 + eta);
			deriv[7 + 16] = 0.1250 * (1 - chsi) * (1 + eta);
			for (int i = 0; i < 4; i++)
				deriv[i + 16] = -deriv[i + 20];

			// get coordinates
			for(int i = 0; i < 8; i++) {
				coords[3*i] = eleNodeCoords[3*numEl*i  + idx];
				coords[3*i + 1] = eleNodeCoords[3*numEl*i + numEl + idx];
				coords[3*i + 2] = eleNodeCoords[3*numEl*i + 2*numEl + idx];
			 }


			// Compute Jacobian
			for (int k = 0; k < 3; k++)
				for (int j = 0; j < 3; j++)
					for (int i = 0; i < 8; i++)
						Jac[k* 3 + j] += deriv[k * 8 + i] * coords[i * 3 + j];


			//determinant of Jacobian
			detJac = Jac[0] * Jac[4] * Jac[8] + Jac[1] * Jac[5] * Jac[6] +
					Jac[3] * Jac[7] * Jac[2] - Jac[2] * Jac[4] * Jac[6] -
					Jac[0] * Jac[5] * Jac[7] - Jac[1] * Jac[3] * Jac[8];

			//Calculate mass matrix
			for (int i = 0; i < 8; i++) {
				int ig = eleNodes[numEl*i + idx];
				for (int j = 0; j < 8; j++)
					globMass[i*nn + ig] += N[i] * rho * cpPoint * N[j] * detJac;
			}
		}
	}
}

__global__ void initializeStiffness(const double* __restrict__ eleMat, const double* __restrict__ eleNodeCoords,
		double* eleStiffness, const int* __restrict__ eleNodes, const int numEl, const int nn) {

	int idx = blockIdx.x*blockDim.x + threadIdx.x;

	if(idx < numEl) {

		double coords[24];

		//shape function

		double chsi;
		double eta;
		double zeta;

		double iJac[9];
		double Jac[9];
		double gradN[24];
		double deriv[24];
		double swap;


		double detJac = 0;

		double kappa = eleMat[5*numEl + idx];

		for(int ip = 0; ip < 8; ++ip) {

			for(int i = 0; i < 24; i++)
				gradN[i] = 0;


			for(int i = 0; i < 9; i++)
				Jac[i] = 0;


			//compute shape function
			chsi = parCoords[ip*3 + 0];
			eta = parCoords[ip*3 + 1];
			zeta = parCoords[ip*3 + 2];

			//compute derivative of shape functions

			// with respect to chsi
			deriv[0] = -0.1250 * (1 - eta) * (1 - zeta);
			deriv[2] =  0.1250 * (1 + eta) * (1 - zeta);
			deriv[4] = -0.1250 * (1 - eta) * (1 + zeta);
			deriv[6] =  0.1250 * (1 + eta) * (1 + zeta);
			for (int i = 0; i < 4; i++)
				deriv[i * 2 + 1] = -deriv[i * 2];

			// with respect to eta
			deriv[0 + 8] = -0.1250 * (1 - chsi) * (1 - zeta);
			deriv[3 + 8] = -deriv[8];
			deriv[1 + 8] = -0.1250 * (1 + chsi) * (1 - zeta);
			deriv[2 + 8] = -deriv[9];
			deriv[4 + 8] = -0.1250 * (1 - chsi) * (1 + zeta);
			deriv[7 + 8] = -deriv[12];
			deriv[5 + 8] = -0.1250 * (1 + chsi) * (1 + zeta);
			deriv[6 + 8] = -deriv[13];

			// with respect to zeta
			deriv[4 + 16] = 0.1250 * (1 - chsi) * (1 - eta);
			deriv[5 + 16] = 0.1250 * (1 + chsi) * (1 - eta);
			deriv[6 + 16] = 0.1250 * (1 + chsi) * (1 + eta);
			deriv[7 + 16] = 0.1250 * (1 - chsi) * (1 + eta);
			for (int i = 0; i < 4; i++)
				deriv[i + 16] = -deriv[i + 20];

			// get coordinates
			for(int i = 0; i < 8; i++) {
				coords[3*i] = eleNodeCoords[3*numEl*i  + idx];
				coords[3*i + 1] = eleNodeCoords[3*numEl*i + numEl + idx];
				coords[3*i + 2] = eleNodeCoords[3*numEl*i + 2*numEl + idx];
			 }


			// Compute Jacobian
			for (int k = 0; k < 3; k++)
				for (int j = 0; j < 3; j++)
					for (int i = 0; i < 8; i++)
						Jac[k* 3 + j] += deriv[k * 8 + i] * coords[i * 3 + j];

			//determinant of Jacobian
			detJac = Jac[0] * Jac[4] * Jac[8] + Jac[1] * Jac[5] * Jac[6] +
					Jac[3] * Jac[7] * Jac[2] - Jac[2] * Jac[4] * Jac[6] -
					Jac[0] * Jac[5] * Jac[7] - Jac[1] * Jac[3] * Jac[8];


			//compute inverse of Jacobian

			iJac[0] = (1/detJac) * Jac[4] * Jac[8] - Jac[5] * Jac[7];
			iJac[1] = (-1/detJac) * Jac[3] * Jac[8] - Jac[5] * Jac[6];
			iJac[2] = (1/detJac) * Jac[3] * Jac[7] - Jac[4] * Jac[6];

			iJac[3] = (-1/detJac) * Jac[1] * Jac[8] - Jac[2] * Jac[7];
			iJac[4] = (1/detJac) * Jac[0] * Jac[8] - Jac[2] * Jac[6];
			iJac[5] = (-1/detJac) * Jac[0] * Jac[7] - Jac[1] * Jac[6];

			iJac[6] = (1/detJac) * Jac[1] * Jac[5] - Jac[2] * Jac[4];
			iJac[7] = (-1/detJac) * Jac[0] * Jac[5] - Jac[2] * Jac[3];
			iJac[8] = (1/detJac) * Jac[0] * Jac[4] - Jac[1] * Jac[3];

			swap = iJac[1];
			iJac[1] = iJac[3];
			iJac[3] = swap;

			swap = iJac[2];
			iJac[2] = iJac[6];
			iJac[6] = swap;

			swap = iJac[5];
			iJac[5] = iJac[7];
			iJac[7] = swap;

			//compute derivative of shape function w.r.t real coordinates


			for (int k = 0; k < 8; k++)
				for (int j = 0; j < 3; j++)
				  for (int i = 0; i < 3; i++)
					gradN[j*8 + k] += iJac[j*3 + i] * deriv[i*8 + k];


			//Calculate element stiffness matrix
			int count = 0;
			for (int i=0; i<8; i++)
			  for (int j = i; j < 8; j++) {
				for (int k = 0; k < 3; k++)
					eleStiffness[count*numEl + idx] += gradN[k * 8 + i] * kappa * gradN[k * 8 + j] * detJac;

				count++;
			  }
		}
	}
}
